#include "hip/hip_runtime.h"
// 在简单矩阵乘法的基础上加入计时的功能，以及错误信息的记录
// 1. 使用hipEventCreate创建事件对象
// 2. 使用hipEventRecord记录事件
// 3. 使用hipEventSynchronize同步事件
// 4. 使用hipEventElapsedTime计算时间
// 5. 使用hipGetErrorString获取错误信息，封装在error.cuh中的CHECK函数中


//简单的矩阵乘法
#include <stdio.h>
#include <math.h>
#include"error.cuh"

//定义block的大小
#define BLOCK_SIZE 32

//
// @brief: cpu矩阵乘法
// @param: a, b, c, size
// @paramDesc: a, b, c分别是两个矩阵和结果矩阵的指针，size是矩阵的大小
// @return: void
//
void cpu_matrix_mult(int *a, int *b, int *c, const int size)
{
    for(int y=0; y<size; ++y)
    {
        for(int x=0; x<size; ++x)
        {
            int tmp = 0;
            //在第x行和第y列进行乘法运算
            //step是第x行和第y列的元素个数
            //作图帮助理解

            //                         b00 b01 b02 b03
            //                         b10 b11 b12 b13
            //                         b20 b21 b22 b23
            //                         b30 b31 b32 b33
            //
            // a00 a01 a02 a03         c00 c01 c02 c03
            // a10 a11 a12 a13         c10 c11 c12 c13    
            // a20 a21 a22 a23         c20 c21 c22 c23
            // a30 a31 a32 a33         c30 c31 c32 c33
            //
            //                          x=2  y=1
            // c21 = a20 * b01 + a21 * b11 + a22 * b21 + a23 * b31
            //          step0      step1       step2       step3

            for(int step = 0; step < size; ++step)
            {
                tmp += a[y*size + step] * b[step * size + x];
            }
            c[y * size + x] = tmp;
        }
    }
}


//
// @brief: gpu矩阵乘法
// @param: a, b, c, size
// @paramDesc: a, b, c分别是两个矩阵和结果矩阵的指针，size是矩阵的大小
// @return: void
//
__global__ void gpu_matrix_mult(int *a, int *b, int *c, const int size)
{
    //计算当前线程的全局坐标
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    int tmp = 0;

    //判断当前线程是否在矩阵范围内
    //容易遗忘的一步
    if( x < size && y < size)
    {
        for( int step = 0; step < size; ++step)
        {
            tmp += a[y * size + step] * b[step * size + x];
        }
        c[y * size + x] = tmp;
    }
}



int main()
{
    //定义矩阵的大小
    int matrix_size = 1000;

    //计算矩阵的内存大小
    int memsize = sizeof(int) * matrix_size * matrix_size;

    //分配内存
    int *h_a, *h_b, *h_c, *h_cc;

    //hipHostMalloc分配的内存是固定内存，可以被cpu和gpu访问
    //谨慎使用固定内存，因为它受限于系统上可用的物理RAM。分配大量的固定内存可能导致内存耗尽问题。
    //使用hipHostFree释放内存
    hipHostMalloc( (void**)&h_a, memsize);
    hipHostMalloc( (void**)&h_b, memsize);
    hipHostMalloc( (void**)&h_c, memsize);
    hipHostMalloc( (void**)&h_cc, memsize);

    //初始化矩阵
    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            h_a[y * matrix_size + x] = rand() % 1024;
        }
    }

    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            h_b[y * matrix_size + x] = rand() % 1024;
        }
    }

    //分配GPU内存
    int *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a , memsize);
    hipMalloc((void**) &d_b , memsize);
    hipMalloc((void**) &d_c , memsize);

    //将数据从cpu内存拷贝到gpu内存
    hipMemcpy( d_a, h_a, memsize, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, memsize, hipMemcpyHostToDevice);

    //定义grid和block的大小
    unsigned int grid_rows = (matrix_size +BLOCK_SIZE -1)/BLOCK_SIZE;
    unsigned int grid_cols = (matrix_size +BLOCK_SIZE -1)/BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);//1.gpu warp 32 2. <= 1024

    //调用hipEventCreate创建事件对象
    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_gpu);
    hipEventCreate(&stop_cpu);

    //记录开始时间
    hipEventRecord(start, 0);
    hipEventSynchronize(start);

    //开始GPU计时
    //调用kernel函数
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, matrix_size);

    //记录GPU结束时间
    hipEventRecord(stop_gpu, 0);
    hipEventSynchronize(stop_gpu);

    //将结果从gpu内存拷贝到cpu内存
    hipMemcpy( h_c, d_c, memsize, hipMemcpyDeviceToHost);

    //开始CPU计时
    hipEventRecord(stop_cpu, 0);
    hipEventSynchronize(stop_cpu);

    //调用cpu函数
    cpu_matrix_mult(h_a, h_b, h_cc, matrix_size);

    //记录CPU结束时间
    hipEventRecord(stop_cpu, 0);
    hipEventSynchronize(stop_cpu);

    //计算时间
    float elapsedTime_gpu, elapsedTime_cpu;
    hipEventElapsedTime(&elapsedTime_gpu, start, stop_gpu);
    hipEventElapsedTime(&elapsedTime_cpu, stop_gpu, stop_cpu);

    //输出时间
    printf("Time to calculate results on GPU: %f ms.\n", elapsedTime_gpu);
    printf("Time to calculate results on CPU: %f ms.\n", elapsedTime_cpu);


    //验证结果
    bool errors = false;
    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            if(fabs(h_cc[y*matrix_size + x] - h_c[y*matrix_size + x]) > (1.0e-10))
            {
                //printf("%d, %d\n", y, x);
                errors = true;
            }
        }
    }
    printf("Result: %s\n", errors?"Errors":"Passed");

    //释放内存
    hipHostFree(h_a );
    hipHostFree(h_b );
    hipHostFree(h_c );
    hipHostFree(h_cc );
    hipFree(d_a );
    hipFree(d_b );
    hipFree(d_c );
    return 0;

}


