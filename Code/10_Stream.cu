#include "hip/hip_runtime.h"
// cuda stream
// cuda lib, cudnn cublas tensort
// 流Stream——一系列的指令执行队列
// mul-stream -- asyn -- order-- asyn异步
// CUDA流也是很多CUDA加速库(cuBLAS, cuDNN, TensorRT)中常用的手段, 它能让多个执行队列并行执行, 还能让这些队列执行的过程中相对独立, 彼此不受影响

#include <stdio.h>
#include <math.h>

// a[] + b[] = c[]

#define N (1024 * 1024)
#define FULL_SIZE (N * 30)

//
// @brief: kernel
// @param: a, b, c
// @return: void
//
__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if( idx < N)
    {
        int idx1 = (idx + 1)%256;
        int idx2 = (idx + 2)%256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0;

        c[idx] = (as + bs)/2;
    }
}

int main()
{
    //获取设备属性
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);

    //检查设备是否支持流
    if( !prop.deviceOverlap )
    {
        printf("Your device will not support speed up from multi-streams\n");
        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t my_stream[3];

    int *h_a, *h_b, *h_c;
    int *d_a0, *d_b0, *d_c0;
    int *d_a1, *d_b1, *d_c1;
    int *d_a2, *d_b2, *d_c2;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStreamCreate(&my_stream[0]);
    hipStreamCreate(&my_stream[1]);
    hipStreamCreate(&my_stream[2]);

    hipMalloc((void**) &d_a0, N * sizeof(int));
    hipMalloc((void**) &d_b0, N * sizeof(int));
    hipMalloc((void**) &d_c0, N * sizeof(int));
    hipMalloc((void**) &d_a1, N * sizeof(int));
    hipMalloc((void**) &d_b1, N * sizeof(int));
    hipMalloc((void**) &d_c1, N * sizeof(int));
    hipMalloc((void**) &d_a2, N * sizeof(int));
    hipMalloc((void**) &d_b2, N * sizeof(int));
    hipMalloc((void**) &d_c2, N * sizeof(int));

    hipHostAlloc((void**) &h_a, FULL_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**) &h_b, FULL_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**) &h_c, FULL_SIZE * sizeof(int), hipHostMallocDefault);


    for(int i = 0; i<FULL_SIZE; i++)
    {
        h_a[i] = rand()%1024;
        h_b[i] = rand()%1024;
    }

    hipEventRecord(start);
    for(int i = 0; i < FULL_SIZE; i += N * 1)
    {
        hipMemcpyAsync(d_a0, h_a+i, N*sizeof(int), hipMemcpyHostToDevice, my_stream[0]);
        hipMemcpyAsync(d_a1, h_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, my_stream[1]);
        //hipMemcpyAsync(d_a2, h_a+i+N+N, N*sizeof(int), hipMemcpyHostToDevice, my_stream[2]);
        hipMemcpyAsync(d_b0, h_a+i, N*sizeof(int), hipMemcpyHostToDevice, my_stream[0]);
        hipMemcpyAsync(d_b1, h_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, my_stream[1]);
        //hipMemcpyAsync(d_b2, h_a+i+N+N, N*sizeof(int), hipMemcpyHostToDevice, my_stream[2]);

        kernel<<<N/256, 256, 0, my_stream[0]>>>(d_a0, d_b0, d_c0);
        kernel<<<N/256, 256, 0, my_stream[1]>>>(d_a1, d_b1, d_c1);
        //kernel<<<N/256, 256, 0, my_stream[2]>>>(d_a2, d_b2, d_c2);

        hipMemcpyAsync(h_c+i, d_c0, N*sizeof(int), hipMemcpyDeviceToHost, my_stream[0]);
        hipMemcpyAsync(h_c+i+N, d_c0, N*sizeof(int), hipMemcpyDeviceToHost, my_stream[0]);
        //hipMemcpyAsync(h_c+i+N+N, d_c0, N*sizeof(int), hipMemcpyDeviceToHost, my_stream[0]);

    }

    hipStreamSynchronize(my_stream[0]);
    hipStreamSynchronize(my_stream[1]);
    hipStreamSynchronize(my_stream[2]);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime , start, stop);

    printf("Time: %3.2f ms\n", elapsedTime);

    // hipFree
    hipFree(d_a0);
    hipFree(d_b0);
    hipFree(d_c0);
    
    return 0;
}
















